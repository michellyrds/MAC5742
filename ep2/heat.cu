#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define WALL_TEMP 20.0
#define FIREPLACE_TEMP 100.0

#define FIREPLACE_START 3
#define FIREPLACE_END 7
#define ROOM_SIZE 10

void initialize(double *h, int n) {
    int fireplace_start = (FIREPLACE_START * n) / ROOM_SIZE;
    int fireplace_end = (FIREPLACE_END * n) / ROOM_SIZE;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == 0 || i == n - 1 || j == 0 || j == n - 1) {
                h[i * n + j] = (i == n - 1 && j >= fireplace_start && j <= fireplace_end) ? FIREPLACE_TEMP : WALL_TEMP;
            } else {
                h[i * n + j] = 0.0;
            }
        }
    }
}

// CUDA
__global__ void jacobi_iteration(double *h, double *g, int n, int iter_limit) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (i < n - 1 && j < n - 1) {
        g[i * n + j] = 0.25 * (h[(i - 1) * n + j] + h[(i + 1) * n + j] + h[i * n + (j - 1)] + h[i * n + (j + 1)]);
        __syncthreads();

        h[i * n + j] = g[i * n + j];
        __syncthreads();
    }
}

double calculate_elapsed_time(struct timespec start, struct timespec end) {
    double start_sec = (double) start.tv_sec * 1e9 + (double) start.tv_nsec;
    double end_sec = (double) end.tv_sec * 1e9 + (double) end.tv_nsec;
    return (end_sec - start_sec) / 1e9;
}

void save_to_file(double *h, int n) {
    FILE *file = fopen("cuda_room.txt", "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            fprintf(file, "%lf ", h[i * n + j]);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

int main(int argc, char *argv[]) {
    if (argc < 5) {
        fprintf(stderr, "Uso: %s <número de pontos> <limite de iterações> <número de threads por bloco> <número de blocos por grade>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int iter_limit = atoi(argv[2]);
    int t = atoi(argv[3]);
    int b = atoi(argv[4]);

    b = sqrt(b);
    int block_size = sqrt(t);
    int grid_dim = (n + block_size - 1) / block_size;

    if (b != grid_dim) {
        fprintf(stderr, "Número de blocos por grade inadequado. Use:\n\n\t %s %d %d %d %d\n\n", argv[0], n, iter_limit, t, grid_dim*grid_dim);
        return 1;
    }

    dim3 threads_per_block = dim3(block_size, block_size);
    dim3 grid_size = dim3(grid_dim, grid_dim);

    // printf("block_size: %d x %d\n", block_size, block_size);
    // printf("grid_size: %d x %d\n", grid_dim, grid_dim);

    double *h, *g;

    hipMallocManaged(&h, n * n * sizeof(double));
    hipMallocManaged(&g, n * n * sizeof(double));

    if (h == NULL || g == NULL) {
        fprintf(stderr, "Erro ao alocar memória para h ou g\n");
        exit(EXIT_FAILURE);
    }

    struct timespec start, end;
    initialize(h, n);
    clock_gettime(CLOCK_MONOTONIC, &start);

    for (int iter = 0; iter < iter_limit; iter++) {
        jacobi_iteration<<<grid_size, threads_per_block>>>(h, g, n, iter_limit);
        hipDeviceSynchronize();
    }

    clock_gettime(CLOCK_MONOTONIC, &end);
    save_to_file(h, n);

    double elapsed_time = calculate_elapsed_time(start, end);
    // printf("Tempo de execução: %.9f segundos\n", elapsed_time);
    FILE *file = fopen("cuda_jacobi_iteration.txt", "a");
    fprintf(file, "%s %d %d %d %d\n", argv[0], n, iter_limit, block_size*block_size, grid_dim*grid_dim);
    fprintf(file, "Tempo de execução: %.9f segundos\n\n", elapsed_time);
    fclose(file);

    hipFree(h);
    hipFree(g);

    return 0;
}