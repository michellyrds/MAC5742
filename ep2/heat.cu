#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define WALL_TEMP 20.0
#define FIREPLACE_TEMP 100.0

#define FIREPLACE_START 3
#define FIREPLACE_END 7
#define ROOM_SIZE 10

void initialize(double ** h, int n){
    int fireplace_start = (FIREPLACE_START * n) / ROOM_SIZE;
    int fireplace_end = (FIREPLACE_END * n) / ROOM_SIZE;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == 0 || i == n - 1 || j == 0 || j == n - 1) {
                h[i][j] = (i == n - 1 && j >= fireplace_start && j <= fireplace_end) ? FIREPLACE_TEMP : WALL_TEMP;
            } else {
                h[i][j] = 0.0;
            }
        }
    }
}

// CUDA
__global__ void jacobi_iteration(double ** h, double ** g, int n, int iter_limit) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < n - 1 && j > 0 && j < n - 1) {
        for (int iter = 0; iter < iter_limit; iter++) {
            g[i][j] = 0.25 * (h[i - 1][j] + h[i + 1][j] + h[i][j - 1] + h[i][j + 1]);
            __syncthreads();

            h[i][j] = g[i][j];
            __syncthreads();
        }
    }
}


double calculate_elapsed_time(struct timespec start, struct timespec end) {
    double start_sec = (double) start.tv_sec * 1e9 + (double) start.tv_nsec;
    double end_sec = (double) end.tv_sec * 1e9 + (double) end.tv_nsec;
    return (end_sec - start_sec) / 1e9;
}

void save_to_file(double ** h, int n) {
    FILE * file = fopen("cuda_room.txt", "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            fprintf(file, "%lf ", h[i][j]);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

int main(int argc, char * argv[]) {
    if (argc < 5) {
        fprintf(stderr, "Uso: %s <número de pontos> <limite de iterações> <número de threads por bloco> <número de blocos por grade>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int iter_limit = atoi(argv[2]);
    int t = atoi(argv[3]);
    int b = atoi(argv[4]);

    int block_size = sqrt(t);

    int b = (n + block_size - 1 )/ block_size;

    dim3 threads_per_block = dim3(block_size, block_size);
    dim3 grid_size = dim3(b, b);
    

    printf("block_size: %d x %d\n", block_size, block_size);
    printf("grid_size: %d x %d\n", b, b);

    double **h, **g;

    hipMallocManaged(&h, n * sizeof(double*));
    hipMallocManaged(&g, n * sizeof(double*));

    if (h == NULL || g == NULL) {
        fprintf(stderr, "Erro ao alocar memória para h ou g\n");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < n; i++) {
        hipMallocManaged(&h[i], n * sizeof(double));
        hipMallocManaged(&g[i], n * sizeof(double));
        if (h[i] == NULL || g[i] == NULL) {
            fprintf(stderr, "Erro ao alocar memória para h[%d] ou g[%d]\n", i, i);
            exit(EXIT_FAILURE);
        }
    }

    struct timespec start, end;
    initialize(h, n);
    clock_gettime(CLOCK_MONOTONIC, &start);
    jacobi_iteration<<<grid_size, threads_per_block>>>(h, g, n, iter_limit);
    clock_gettime(CLOCK_MONOTONIC, &end);
    hipDeviceSynchronize();
    save_to_file(h, n);

    double elapsed_time = calculate_elapsed_time(start, end);
    printf("Tempo de execução: %.9f segundos\n", elapsed_time);

    for (int i = 0; i < n; i++) {
        hipFree(h[i]);
        hipFree(g[i]);
    }
    hipFree(h);
    hipFree(g);

    return 0;
}