#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define WALL_TEMP 20.0
#define FIREPLACE_TEMP 100.0
#define BODY_TEMP 37.0

#define FIREPLACE_START 3
#define FIREPLACE_END 7
#define ROOM_SIZE 10

#define BODY_START_X 4
#define BODY_END_X 6
#define BODY_START_Y 4
#define BODY_END_Y 6

void initialize(double *h, int n) {
    int fireplace_start = (FIREPLACE_START * n) / ROOM_SIZE;
    int fireplace_end = (FIREPLACE_END * n) / ROOM_SIZE;
    int body_start_x = (BODY_START_X * n) / ROOM_SIZE;
    int body_end_x = (BODY_END_X * n) / ROOM_SIZE;
    int body_start_y = (BODY_START_Y * n) / ROOM_SIZE;
    int body_end_y = (BODY_END_Y * n) / ROOM_SIZE;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == 0 || i == n - 1 || j == 0 || j == n - 1) {
                h[i * n + j] = (i == n - 1 && j >= fireplace_start && j <= fireplace_end) ? FIREPLACE_TEMP : WALL_TEMP;
            } else if (i >= body_start_x && i <= body_end_x && j >= body_start_y && j <= body_end_y) {
                h[i * n + j] = BODY_TEMP;
            } else {
                h[i * n + j] = 0.0;
            }
        }
    }
}

// CUDA
__global__ void jacobi_iteration(double *h, double *g, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    int body_start_x = (BODY_START_X * n) / ROOM_SIZE;
    int body_end_x = (BODY_END_X * n) / ROOM_SIZE;
    int body_start_y = (BODY_START_Y * n) / ROOM_SIZE;
    int body_end_y = (BODY_END_Y * n) / ROOM_SIZE;

    if (i < n - 1 && j < n - 1) {
        if (!(i >= body_start_x && i <= body_end_x && j >= body_start_y && j <= body_end_y)) {
            g[i * n + j] = 0.25 * (h[(i - 1) * n + j] + h[(i + 1) * n + j] + h[i * n + (j - 1)] + h[i * n + (j + 1)]);
        }
        __syncthreads();

        if (!(i >= body_start_x && i <= body_end_x && j >= body_start_y && j <= body_end_y)) {
            h[i * n + j] = g[i * n + j];
        }
        __syncthreads();
    }
}

double calculate_elapsed_time(struct timespec start, struct timespec end) {
    double start_sec = (double) start.tv_sec * 1e9 + (double) start.tv_nsec;
    double end_sec = (double) end.tv_sec * 1e9 + (double) end.tv_nsec;
    return (end_sec - start_sec) / 1e9;
}

void save_to_file(double *h, int n) {
    FILE *file = fopen("cuda_room_37.txt", "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            fprintf(file, "%lf ", h[i * n + j]);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

bool is_perfect_square(int n){
    if (ceil((double)sqrt(n)) == floor((double)sqrt(n))) {
        return true;
    }
    return false;
}

int main(int argc, char *argv[]) {
    if (argc < 5) {
        fprintf(stderr, "Uso: %s <número de pontos> <limite de iterações> <número de threads por bloco> <número de blocos por grade>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int iter_limit = atoi(argv[2]);
    int t = atoi(argv[3]);
    int b = atoi(argv[4]);

    int block_size = sqrt(t);
    int grid_dim = (n + block_size - 1) / block_size;

    b = sqrt(b);
    if (b != grid_dim) {
        fprintf(stderr, "Número de blocos por grade inadequado. Use:\n\n\t %s %d %d %d %d\n\n", argv[0], n, iter_limit, t, grid_dim*grid_dim);
        return 1;
    }

    dim3 threads_per_block = dim3(block_size, block_size);
    dim3 grid_size = dim3(grid_dim, grid_dim);

    // printf("block_size: %d x %d\n", block_size, block_size);
    // printf("grid_size: %d x %d\n", grid_dim, grid_dim);

    double *h, *g;

    hipMallocManaged(&h, n * n * sizeof(double));
    hipMallocManaged(&g, n * n * sizeof(double));

    if (h == NULL || g == NULL) {
        fprintf(stderr, "Erro ao alocar memória para h ou g\n");
        exit(EXIT_FAILURE);
    }

    struct timespec start, end;
    initialize(h, n);
    clock_gettime(CLOCK_MONOTONIC, &start);

    for (int iter = 0; iter < iter_limit; iter++) {
        jacobi_iteration<<<grid_size, threads_per_block>>>(h, g, n);
        hipDeviceSynchronize();
    }

    clock_gettime(CLOCK_MONOTONIC, &end);
    save_to_file(h, n);

    double elapsed_time = calculate_elapsed_time(start, end);
    // printf("Tempo de execução: %.9f segundos\n", elapsed_time);
    FILE *file = fopen("cuda_jacobi_iteration_37.txt", "a");
    fprintf(file, "%s %d %d %d %d\n", argv[0], n, iter_limit, block_size*block_size, grid_dim*grid_dim);
    fprintf(file, "Tempo de execução: %.9f segundos\n\n", elapsed_time);
    fclose(file);

    hipFree(h);
    hipFree(g);

    return 0;
}